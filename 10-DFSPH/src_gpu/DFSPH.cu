#include "hip/hip_runtime.h"
#include "DFSPH.cuh"
#include <hip/hip_runtime.h>
#include "common/hip/hip_runtime_api.h"
#include "neighbor/include/cuNSearch.h"
#include "neighbor/src/cuNSearchDeviceData.h"

__constant__ float KERNEL_RADIUS;
__constant__ float KERNEL_K;
__constant__ float KERNEL_L;
static inline __device__ float dot(const float3 a, const float3 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
static inline __device__ float length(const float3 r) { return sqrtf(dot(r, r)); }
static inline __device__ float W(const float3 r)
{
	float res = 0.f;
	const float q = length(r) / KERNEL_RADIUS;
	if (q <= 1.f)
	{
		if (q <= 0.5f)
		{
			const float q2 = q * q;
			const float q3 = q2 * q;
			res = KERNEL_K * (6.f * q3 - 6.f * q2 + 1.f);
		} else
		{
			res = KERNEL_K * (2.f * powf(1.f - q, 3.f));
		}
	}
	return res;
}
static inline __device__ float3 gradW(const float3 r)
{
	float3 res = {0.f, 0.f, 0.f};
	const float rl = length(r);
	const float q = rl / KERNEL_RADIUS;
	if ((rl > 1.0e-9) && (q <= 1.f))
	{
		float3 gradq = {r.x / rl, r.y / rl, r.z / rl};
		gradq.x /= KERNEL_RADIUS;
		gradq.y /= KERNEL_RADIUS;
		gradq.z /= KERNEL_RADIUS;
		if (q <= 0.5f)
		{
			res.x = KERNEL_L * q * (3.f * q - 2.f) * gradq.x;
			res.y = KERNEL_L * q * (3.f * q - 2.f) * gradq.y;
			res.z = KERNEL_L * q * (3.f * q - 2.f) * gradq.z;
		} else
		{
			const float factor = 1.f - q;
			res.x = KERNEL_L * (-factor * factor) * gradq.x;
			res.y = KERNEL_L * (-factor * factor) * gradq.y;
			res.z = KERNEL_L * (-factor * factor) * gradq.z;
		}
	}
	return res;
}
static inline __device__ float W0() { return W(make_float3(0.f, 0.f, 0.f)); }

template<class Func>
__global__ void parallel_for(int n, Func func) { for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) func(i); }

HinaPE::CUDA::DFSPH::DFSPH(float _kernel_radius) : kernel_radius(_kernel_radius), size(0)
{
	float k = 8.f / (3.14159265358979323846f * kernel_radius * kernel_radius * kernel_radius);
	float l = 48.f / (3.14159265358979323846f * kernel_radius * kernel_radius * kernel_radius);
	hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_RADIUS), &kernel_radius, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_K), &k, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_L), &l, sizeof(float));

	Fluid = std::make_shared<FluidGPU>();
	Searcher = std::make_shared<cuNSearch::NeighborhoodSearch>(kernel_radius);
	fluid_idx = Searcher->add_point_set(&(Fluid->x.data()->x), Fluid->x.size(), true, true, true);
}

void HinaPE::CUDA::DFSPH::resize(size_t n)
{
	Fluid->x.resize(n);
	Fluid->v.resize(n);
	Fluid->a.resize(n);
	Fluid->m.resize(n);
	Fluid->V.resize(n);
	Fluid->rho.resize(n);
	size = n;
}

void HinaPE::CUDA::DFSPH::solve(float dt)
{
	Searcher->update_point_set(fluid_idx);
	Searcher->find_neighbors();
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					a = Fluid->a.data(),
					m = Fluid->m.data(),
					V = Fluid->V.data(),
					rho = Fluid->rho.data()
			] __device__(size_t i)
			{
				// compute density
				float rho_i = V[i] * W0();
			});
}

#ifdef TEST_DFSPH
int main()
{
	int n = 65536;
	HinaPE::CUDA::DFSPH df(0.04f);
	df.resize(n);
	df.solve(0.02f);
	return 0;
}
#endif
