#include "hip/hip_runtime.h"
#include "DFSPH.cuh"
#include <hip/hip_runtime.h>
#include "common/hip/hip_runtime_api.h"
#include "common/hip/hip_vector_types.h"
#include "neighbor/include/cuNSearch.h"
#include "neighbor/src/cuNSearchDeviceData.h"
#include <thrust/execution_policy.h>

__constant__ float PARTICLE_RADIUS;
__constant__ float KERNEL_RADIUS;
__constant__ float KERNEL_K;
__constant__ float KERNEL_L;
__constant__ float REST_DENSITY;
__constant__ float3 MAX_BOUND;
__constant__ bool TOP_OPEN;
static inline __device__ float W(const float3 r)
{
	float res = 0.f;
	const float q = length(r) / KERNEL_RADIUS;
	if (q <= 1.f)
	{
		if (q <= 0.5f)
		{
			const float q2 = q * q;
			const float q3 = q2 * q;
			res = KERNEL_K * (6.f * q3 - 6.f * q2 + 1.f);
		} else
		{
			res = KERNEL_K * (2.f * powf(1.f - q, 3.f));
		}
	}
	return res;
}
static inline __device__ float3 gradW(const float3 r)
{
	float3 res = {0.f, 0.f, 0.f};
	const float rl = length(r);
	const float q = rl / KERNEL_RADIUS;
	if ((rl > 1.0e-9) && (q <= 1.f))
	{
		float3 gradq = {r.x / rl, r.y / rl, r.z / rl};
		gradq.x /= KERNEL_RADIUS;
		gradq.y /= KERNEL_RADIUS;
		gradq.z /= KERNEL_RADIUS;
		if (q <= 0.5f)
		{
			res.x = KERNEL_L * q * (3.f * q - 2.f) * gradq.x;
			res.y = KERNEL_L * q * (3.f * q - 2.f) * gradq.y;
			res.z = KERNEL_L * q * (3.f * q - 2.f) * gradq.z;
		} else
		{
			const float factor = 1.f - q;
			res.x = KERNEL_L * (-factor * factor) * gradq.x;
			res.y = KERNEL_L * (-factor * factor) * gradq.y;
			res.z = KERNEL_L * (-factor * factor) * gradq.z;
		}
	}
	return res;
}
static inline __device__ float W0() { return W(make_float3(0.f, 0.f, 0.f)); }

HinaPE::CUDA::DFSPH::DFSPH(float _kernel_radius) : size(0)
{
	float kr = _kernel_radius;
	float r = 0.02f;
	float k = 8.f / (3.14159265358979323846f * kr * kr * kr);
	float l = 48.f / (3.14159265358979323846f * kr * kr * kr);
	float rd = 1000.f;
	float3 max_bound = {1.f, 1.f, 1.f};
	bool top_open = true;
	hipMalloc((void **) &PARTICLE_RADIUS, sizeof(float));
	hipMalloc((void **) &KERNEL_RADIUS, sizeof(float));
	hipMalloc((void **) &KERNEL_K, sizeof(float));
	hipMalloc((void **) &KERNEL_L, sizeof(float));
	hipMalloc((void **) &REST_DENSITY, sizeof(float));
	hipMalloc((void **) &MAX_BOUND, sizeof(float3));
	hipMalloc((void **) &TOP_OPEN, sizeof(bool));
	hipMemcpyToSymbol(HIP_SYMBOL(PARTICLE_RADIUS), &r, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_RADIUS), &kr, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_K), &k, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_L), &l, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(REST_DENSITY), &rd, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(MAX_BOUND), &max_bound, 3 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(TOP_OPEN), &top_open, sizeof(bool));

	Fluid = std::make_shared<FluidGPU>();
	Searcher = std::make_shared<cuNSearch::NeighborhoodSearch>(kr);
	fluid_idx = Searcher->add_point_set(&(Fluid->x.data()->x), Fluid->x.size(), true, true, true);
	Searcher->set_active(true);
}

void HinaPE::CUDA::DFSPH::resize(size_t n)
{
	Fluid->x.resize(n);
	Fluid->v.resize(n);
	Fluid->a.resize(n);
	Fluid->m.resize(n);
	Fluid->V.resize(n);
	Fluid->rho.resize(n);
	size = n;
}

void HinaPE::CUDA::DFSPH::solve(float dt)
{

	// ==================== 1. Build Neighbors ====================
	Searcher->update_point_set(fluid_idx);
	Searcher->find_neighbors();
	cuNSearch::PointSet::NeighborSet &neighbor_set = Searcher->point_set(fluid_idx).get_raw_neighbor_set(fluid_idx);



	// ==================== 2. Compute Density and Factor ====================
	thrust::for_each( // compute density and factor
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					V = Fluid->V.data(),
					rho = Fluid->rho.data(),
					factor = Fluid->factor.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
			] __device__(size_t i)
			{
				float _rho_i = V[i] * W0();
				float _sum_grad_p_k = 0;
				float3 _grad_p_i = {0.f, 0.f, 0.f};
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];
					_rho_i += V[j] * W(x[i] - x[j]);

					float3 _grad_p_j = -V[j] * gradW(x[i] - x[j]);
					_sum_grad_p_k += dot(_grad_p_j, _grad_p_j);
					_grad_p_i -= _grad_p_j;
				}

				_sum_grad_p_k += dot(_grad_p_i, _grad_p_i);
				rho[i] = _rho_i * REST_DENSITY;
				if (_sum_grad_p_k > 1e-6f)
					factor[i] = -1.f / _sum_grad_p_k;
				else
					factor[i] = 0;
			});



	// ==================== 3. Divergence Solve ====================
	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
	_) { return _ / dt; });
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					V = Fluid->V.data(),
					density_adv = Fluid->density_adv.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
			] __device__(size_t i)
			{
				float _d_adv = 0;
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];
					_d_adv += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
				}
				density_adv[i] = max(_d_adv, 0.f);
			});
	uint iteration_v = 0;
	while (iteration_v < 1 || iteration_v < 100)
	{
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						factor = Fluid->factor.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
						dt
				] __device__(size_t i)
				{
					float _k_i = density_adv[i] * factor[i];
					float3 _dv = {0.f, 0.f, 0.f};
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						float _k_j = density_adv[j] * factor[j];
						float _k_sum = _k_i + _k_j;
						if (_k_sum > 1e-5f || _k_sum < -1e-5f)
						{
							float3 _grad_p_j = -V[j] * gradW(x[i] - x[j]);
							_dv -= dt * _k_sum * _grad_p_j;
						}
					}
					v[i] += _dv;
				});
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
				] __device__(size_t i)
				{
					float _d_adv = 0;
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						_d_adv += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
					}
					density_adv[i] = max(_d_adv, 0.f);
				});
		thrust::transform(Fluid->density_adv.begin(), Fluid->density_adv.end(), Fluid->density_adv.begin(), [] __device__(float
																														  _) { return _ * REST_DENSITY; });
		float avg_density_error = thrust::reduce(Fluid->density_adv.begin(), Fluid->density_adv.end(), 0.f, thrust::plus<float>()) / size;
		if (avg_density_error < 1.f)
			break;
		++iteration_v;
	}
	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
	_) { return _ * dt; });



	// ==================== 4. Non-Pressure Force and Predict Velocity ====================
	thrust::transform(Fluid->a.begin(), Fluid->a.end(), Fluid->a.begin(), [] __device__(float3) { return make_float3(0, -9.8f, 0); });
	thrust::transform(Fluid->a.begin(), Fluid->a.end(), Fluid->v.begin(), Fluid->v.begin(),[dt] __device__(float3
	a, float3
	v) { return v + dt * a; });



	// ==================== 5. Pressure Solve ====================
	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
	_) { return _ / (dt * dt); });
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					V = Fluid->V.data(),
					rho = Fluid->rho.data(),
					density_adv = Fluid->density_adv.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
			] __device__(size_t i)
			{
				float _delta = 0;
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];
					_delta += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
				}
				float _d_adv = rho[i] / REST_DENSITY * _delta;
				density_adv[i] = max(_d_adv, 1.f);
			});
	uint iteration_d = 0;
	while (iteration_d < 1 || iteration_d < 100)
	{
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						factor = Fluid->factor.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
						dt
				] __device__(size_t i)
				{
					float _k_i = (density_adv[i] - 1.f) * factor[i];
					float3 _dv = {0.f, 0.f, 0.f};
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						float _k_j = (density_adv[j] - 1.f) * factor[j];
						float _k_sum = _k_i + _k_j;
						if (_k_sum > 1e-5f || _k_sum < -1e-5f)
						{
							float3 _grad_p_j = -V[j] * gradW(x[i] - x[j]);
							_dv -= dt * _k_sum * _grad_p_j;
						}
					}
					v[i] += _dv;
				});
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						rho = Fluid->rho.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
				] __device__(size_t i)
				{
					float _delta = 0;
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						_delta += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
					}
					float _d_adv = rho[i] / REST_DENSITY * _delta;
					density_adv[i] = max(_d_adv, 1.f);
				});
		thrust::transform(Fluid->density_adv.begin(), Fluid->density_adv.end(), Fluid->density_adv.begin(), [] __device__(float _) { return (_ - 1.f) * REST_DENSITY; });
		float avg_density_error = thrust::reduce(Fluid->density_adv.begin(), Fluid->density_adv.end(), 0.f, thrust::plus<float>()) / size;
		if (avg_density_error < 1.f)
			break;
		++iteration_d;
	}
	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
	_) { return _ * (dt * dt); });



	// ==================== 6. Advection ====================
	thrust::transform(Fluid->v.begin(), Fluid->v.end(), Fluid->x.begin(), Fluid->x.begin(),[dt] __device__(float3
	v, float3
	x) { return x + dt * v; });



	// ==================== 6. Boundary ====================
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data()
			] __device__(size_t i)
			{
				float3 collision_normal{0, 0, 0};
				if (x[i].x > MAX_BOUND.x)
				{
					x[i].x = MAX_BOUND.x;
					collision_normal.x += 1;
				}
				if (x[i].x < -MAX_BOUND.x)
				{
					x[i].x = -MAX_BOUND.x;
					collision_normal.x -= 1;
				}
				if (!TOP_OPEN)
				{
					if (x[i].y > MAX_BOUND.y)
					{
						x[i].y = MAX_BOUND.y;
						collision_normal.y += 1;
					}
				}
				if (x[i].y < -MAX_BOUND.y)
				{
					x[i].y = -MAX_BOUND.y;
					collision_normal.y -= 1;
				}
				if (x[i].z > MAX_BOUND.z)
				{
					x[i].z = MAX_BOUND.z;
					collision_normal.z += 1;
				}
				if (x[i].z < -MAX_BOUND.z)
				{
					x[i].z = -MAX_BOUND.z;
					collision_normal.z -= 1;
				}
				collision_normal = normalize(collision_normal);
				v[i] -= (1. + 0.5f) * dot(v[i], collision_normal) * collision_normal;
			});
}
void HinaPE::CUDA::DFSPH::solve_test(float dt)
{

}

#ifdef TEST_DFSPH
#include <vector>
#include <numeric>
int main()
{
	using Real = float;
	using Real3 = float3;
	std::vector<Real3> positions;
	std::size_t const N = 120;
	Real const r_omega = static_cast<Real>(0.15);
	Real const r_omega2 = r_omega * r_omega;
	Real const radius = static_cast<Real>(2.0) * (static_cast<Real>(2.0) * r_omega / static_cast<Real>(N - 1));

//Generate test data
	Real min_x = std::numeric_limits<Real>::max();
	Real max_x = std::numeric_limits<Real>::min();
	positions.reserve(N * N * N);
	for (uint i = 0; i < N; ++i)
	{
		for (uint j = 0; j < N; ++j)
		{
			for (uint k = 0; k < N; ++k)
			{
				std::array<Real, 3> x = {{
												 r_omega * static_cast<Real>(2.0 * static_cast<double>(i) / static_cast<double>(N - 1) - 1.0),
												 r_omega * static_cast<Real>(2.0 * static_cast<double>(j) / static_cast<double>(N - 1) - 1.0),
												 r_omega * static_cast<Real>(2.0 * static_cast<double>(k) / static_cast<double>(N - 1) - 1.0)}};

				Real l2 = x[0] * x[0] + x[1] * x[1] + x[2] * x[2];
				if (l2 < r_omega2)
				{
					x[0] += static_cast<Real>(0.35);
					x[1] += static_cast<Real>(0.35);
					x[2] += static_cast<Real>(0.35);
					positions.push_back(make_float3(x[0], x[1], x[2]));
					if (min_x > x[0])
					{
						min_x = x[0];
					}
					if (max_x < x[0])
					{
						max_x = x[0];
					}
				}
			}
		}
	}
	printf("Number of particles: %d \n", static_cast<int>(positions.size()));

	//Create neighborhood search instance
	cuNSearch::NeighborhoodSearch nsearch(radius);

	//Add point set from the test data
	auto pointSetIndex = nsearch.add_point_set(&positions.front().x, positions.size(), true, true);

	for (size_t i = 0; i < 5; i++)
	{
		if (i != 0)
		{
			nsearch.z_sort();
			nsearch.point_set(pointSetIndex).sort_field((Real3 *) nsearch.point_set(pointSetIndex).GetPoints());
		}

		nsearch.find_neighbors();
	}

	//Neighborhood search result test
	auto &pointSet = nsearch.point_set(0);
	auto points = pointSet.GetPoints();

	std::cout << "Validate results" << std::endl;
	for (uint i = 0; i < pointSet.n_points(); i++)
	{
		Real3 point = ((Real3 *) points)[i];
		auto count = pointSet.n_neighbors(0, i);
		for (uint j = 0; j < count; j++)
		{
			auto neighbor = pointSet.neighbor(0, i, j);
			Real3 diff = {point.x - ((Real3 *) points)[neighbor].x, point.y - ((Real3 *) points)[neighbor].y, point.z - ((Real3 *) points)[neighbor].z};
			float squaredLength = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
			float distance = sqrt(squaredLength);

			if (distance > radius)
			{
				throw std::runtime_error("Not a neighbor");
			}
		}
	}
	return 0;
}
// thrust::reduce(thrust::device, rho.get(), rho.get() + 1, 0.f, thrust::plus<float>());
#endif
