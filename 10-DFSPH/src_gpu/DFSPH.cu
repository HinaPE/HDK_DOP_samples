#include "hip/hip_runtime.h"
#include "DFSPH.cuh"
#include <hip/hip_runtime.h>
#include "common/hip/hip_runtime_api.h"
#include "common/hip/hip_vector_types.h"
#include "neighbor/include/cuNSearch.h"
#include "neighbor/src/cuNSearchDeviceData.h"
#include <thrust/execution_policy.h>

__constant__ float GPU_PARTICLE_RADIUS;
__constant__ float GPU_KERNEL_RADIUS;
__constant__ float GPU_KERNEL_K;
__constant__ float GPU_KERNEL_L;
__constant__ float GPU_REST_DENSITY;
__constant__ float GPU_VISCOSITY;
__constant__ float GPU_SURFACE_TENSION;
__constant__ float3 GPU_MAX_BOUND;
__constant__ bool GPU_TOP_OPEN;
static inline __device__ float W(const float r)
{
	float res = 0.f;
	const float q = r / GPU_KERNEL_RADIUS;
	if (q <= 1.f)
	{
		if (q <= 0.5f)
		{
			const float q2 = q * q;
			const float q3 = q2 * q;
			res = GPU_KERNEL_K * (6.f * q3 - 6.f * q2 + 1.f);
		} else
		{
			res = GPU_KERNEL_K * (2.f * powf(1.f - q, 3.f));
		}
	}
	return res;
}
static inline __device__ float W(const float3 r) { return W(length(r)); }
static inline __device__ float3 gradW(const float3 r)
{
	float3 res;
	const float rl = length(r);
	const float q = rl / GPU_KERNEL_RADIUS;
	if ((rl > 1.0e-9) && (q <= 1.f))
	{
		float3 gradq = {r.x / rl, r.y / rl, r.z / rl};
		gradq.x /= GPU_KERNEL_RADIUS;
		gradq.y /= GPU_KERNEL_RADIUS;
		gradq.z /= GPU_KERNEL_RADIUS;
		if (q <= 0.5f)
		{
			res = GPU_KERNEL_L * q * (3.f * q - 2.f) * gradq;
		} else
		{
			const float factor = 1.f - q;
			res = GPU_KERNEL_L * (-factor * factor) * gradq;
		}
	} else
		res = {0.f, 0.f, 0.f};
	return res;
}
static inline __device__ float W_zero() { return W(make_float3(0.f, 0.f, 0.f)); }

HinaPE::CUDA::DFSPH::DFSPH() : size(0)
{
	Fluid = std::make_shared<FluidGPU>();
	Searcher = std::make_shared<cuNSearch::NeighborhoodSearch>(KERNEL_RADIUS);
	fluid_idx = Searcher->add_point_set(&(Fluid->x.data()->x), Fluid->x.size(), true, true, true);
	Searcher->set_active(true);

	hipMalloc((void **) &GPU_PARTICLE_RADIUS, sizeof(float));
	hipMalloc((void **) &GPU_KERNEL_RADIUS, sizeof(float));
	hipMalloc((void **) &GPU_KERNEL_K, sizeof(float));
	hipMalloc((void **) &GPU_KERNEL_L, sizeof(float));
	hipMalloc((void **) &GPU_REST_DENSITY, sizeof(float));
	hipMalloc((void **) &GPU_VISCOSITY, sizeof(float));
	hipMalloc((void **) &GPU_SURFACE_TENSION, sizeof(float));
	hipMalloc((void **) &GPU_MAX_BOUND, sizeof(float3));
	hipMalloc((void **) &GPU_TOP_OPEN, sizeof(bool));

	set_gpu_constants();
}

void HinaPE::CUDA::DFSPH::set_gpu_constants()
{
	float k = 8.f / (3.14159265358979323846f * KERNEL_RADIUS * KERNEL_RADIUS * KERNEL_RADIUS);
	float l = 48.f / (3.14159265358979323846f * KERNEL_RADIUS * KERNEL_RADIUS * KERNEL_RADIUS);
	hipMemcpyToSymbol(HIP_SYMBOL(GPU_PARTICLE_RADIUS), &PARTICLE_RADIUS, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(GPU_KERNEL_RADIUS), &KERNEL_RADIUS, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(GPU_KERNEL_K), &k, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(GPU_KERNEL_L), &l, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(GPU_REST_DENSITY), &REST_DENSITY, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(GPU_SURFACE_TENSION), &SURFACE_TENSION, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(GPU_VISCOSITY), &VISCOSITY, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(GPU_MAX_BOUND), &MaxBound, 3 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(GPU_TOP_OPEN), &TOP_OPEN, sizeof(bool));
}

void HinaPE::CUDA::DFSPH::resize(size_t n)
{
	// ==================== 1. Refresh Fluid Data ====================
	if (size < n)
	{
		auto new_size = n;
		Fluid->x.resize(new_size, {0, 0, 0}); // NO NEED TO RESIZE
		Fluid->v.resize(new_size, {0, 0, 0});
		Fluid->a.resize(new_size, {0, 0, 0});
		Fluid->m.resize(new_size, DEFAULT_M);
		Fluid->V.resize(new_size, DEFAULT_V);
		Fluid->rho.resize(new_size, 0);
		Fluid->factor.resize(new_size, 0);
		Fluid->density_adv.resize(new_size, 0);
		Fluid->nn.resize(new_size, 0);
		Fluid->tmp.resize(new_size, 0);
		size = new_size;

		Searcher->resize_point_set(fluid_idx, &(Fluid->x.data()->x), size);
	}
}

void HinaPE::CUDA::DFSPH::solve(float dt)
{
	// ==================== 2. Build Neighbors ====================
	Searcher->update_point_set(fluid_idx);
	Searcher->find_neighbors();
	cuNSearch::PointSet::NeighborSet &neighbor_set = Searcher->point_set(fluid_idx).get_raw_neighbor_set(fluid_idx);
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					nn = Fluid->nn.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
			] __device__(size_t i)
			{
				nn[i] = dCounts[i];
			});



	// ==================== 3. Compute Density and Factor ====================
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					V = Fluid->V.data(),
					rho = Fluid->rho.data(),
					factor = Fluid->factor.data(),
					nn = Fluid->nn.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
			] __device__(size_t i)
			{
				float sum_grad = 0.;
				float3 grad_i{0, 0, 0};
				rho[i] = V[i] * W_zero();
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];
					rho[i] += V[j] * W(x[i] - x[j]);
					float3 grad_j = -V[j] * gradW(x[i] - x[j]);
					sum_grad += dot(grad_j, grad_j);
					grad_i -= grad_j;
				}
				rho[i] *= GPU_REST_DENSITY;
				sum_grad += dot(grad_i, grad_i);
				if (sum_grad > 1e-6)
					factor[i] = -1.f / sum_grad;
				else
					factor[i] = 0;
			});



	// ==================== 4. Divergence Solver ====================
	thrust::for_each( // compute density change
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					V = Fluid->V.data(),
					density_adv = Fluid->density_adv.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
			] __device__(size_t i)
			{
				float _density_adv = 0;
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];
					_density_adv += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
				}
				density_adv[i] = std::max(_density_adv, 0.f);
			});
	DIVERGENCE_ITERS = 0;
	float avg_density_err = 0.0;
	while (DIVERGENCE_ITERS < 1 || DIVERGENCE_ITERS < MAX_ITERATIONS)
	{
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						factor = Fluid->factor.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
						dt
				] __device__(size_t i)
				{
					float b_i = density_adv[i];
					float k_i = b_i * factor[i] / dt;
					float3 dv{0, 0, 0};
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						float b_j = density_adv[j];
						float k_j = b_j * factor[j] / dt;
						float k_sum = k_i + k_j;
						if (std::abs(k_sum) > 1e-5)
						{
							float3 grad_j = -V[j] * gradW(x[i] - x[j]);
							dv -= dt * k_sum * grad_j;
						}
					}
					v[i] += dv;
				});
		thrust::for_each( // compute density change
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
				] __device__(size_t i)
				{
					float _density_adv = 0;
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						_density_adv += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
					}
					density_adv[i] = std::max(_density_adv, 0.f);
				});
		avg_density_err = 0;
		for (size_t i = 0; i < size; ++i)
			avg_density_err += REST_DENSITY * Fluid->density_adv[i];
		avg_density_err /= size;

		float eta = 1.f / dt * .1f * 0.01f * REST_DENSITY;
		if (avg_density_err <= eta)
			break;
		++DIVERGENCE_ITERS;
	}



	// ==================== 5. Non-Pressure Force and Predict Velocity ====================
	constexpr float d = 10;
	const float diameter = 2 * PARTICLE_RADIUS;
	const float diameter2 = diameter * diameter;
	const float kr2 = KERNEL_RADIUS * KERNEL_RADIUS;
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					V = Fluid->V.data(),
					m = Fluid->m.data(),
					a = Fluid->a.data(),
					rho = Fluid->rho.data(),
					density_adv = Fluid->density_adv.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
					diameter,
					diameter2,
					kr2
			] __device__(size_t i)
			{
				float3 dv{0, -9.8f, 0};
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];

					// Surface Tension
					const float3 r = x[i] - x[j];
					const float r2 = dot(r, r);
					const float r1 = std::sqrt(r2);
					if (r2 > diameter2)
						dv -= GPU_SURFACE_TENSION / m[i] * m[j] * r * W(r1);
					else
						dv -= GPU_SURFACE_TENSION / m[i] * m[j] * r * W(diameter);

					// Fluid Viscosity
					float v_xy = dot(v[i] - v[j], r);
					float3 f_v = d * GPU_VISCOSITY * (m[j] / (rho[j])) * v_xy / (r2 + 0.01f * kr2) * gradW(r);
					dv += f_v;
				}
				a[i] = dv;
			});
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					v = Fluid->v.data(),
					a = Fluid->a.data(),
					dt
			] __device__(size_t i)
			{
				v[i] += dt * a[i];
			});



	// ==================== 6. Pressure Solver ====================
	thrust::for_each( // compute density adv
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					V = Fluid->V.data(),
					rho = Fluid->rho.data(),
					density_adv = Fluid->density_adv.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
					dt
			] __device__(size_t i)
	{
		float delta = 0;
		for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
		{
			uint j = dNeighbors[dOffsets[i] + _n_idx];
			delta += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
		}
		float _density_adv = rho[i] / GPU_REST_DENSITY + dt * delta;
		density_adv[i] = std::max(_density_adv, 1.f);
	});
	PRESSURE_ITERS = 0;
	avg_density_err = 0.0;
	while (PRESSURE_ITERS < 1 || PRESSURE_ITERS < MAX_ITERATIONS)
	{
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						factor = Fluid->factor.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
						dt
				] __device__(size_t i)
		{
			float b_i = density_adv[i] - 1.f;
			float k_i = b_i * factor[i] / (dt * dt);
			float3 dv{0, 0, 0};
			for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
			{
				uint j = dNeighbors[dOffsets[i] + _n_idx];
				float b_j = density_adv[j] - 1.f;
				float k_j = b_j * factor[j] / (dt * dt);
				float k_sum = k_i + k_j;
				if (std::abs(k_sum) > 1e-5)
				{
					float3 grad_p_j = -V[j] * gradW(x[i] - x[j]);
					dv -= dt * k_sum * grad_p_j;
				}
			}
			v[i] += dv;
		});
		thrust::for_each( // compute density adv
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						rho = Fluid->rho.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
						dt
				] __device__(size_t i)
				{
					float delta = 0;
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						delta += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
					}
					float _density_adv = rho[i] / GPU_REST_DENSITY + dt * delta;
					density_adv[i] = std::max(_density_adv, 1.f);
				});
		avg_density_err = 0;
		for (size_t i = 0; i < size; ++i)
			avg_density_err += REST_DENSITY * (Fluid->density_adv[i] - 1.f);
		avg_density_err /= size;

		float eta = 0.05f * 0.01f * REST_DENSITY;
		if (avg_density_err <= eta)
			break;
		++PRESSURE_ITERS;
	}



	// ==================== 7. Advect ====================
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					dt
			] __device__(size_t i)
	{
		x[i] += dt * v[i];
	});



	// ==================== 8. Enforce Boundary ====================
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data()
			] __device__(size_t i)
	{
		float3 normal{0, 0, 0};
		if (x[i].x > GPU_MAX_BOUND.x)
		{
			x[i].x = GPU_MAX_BOUND.x;
			normal.x += 1;
		}
		if (x[i].x < -GPU_MAX_BOUND.x)
		{
			x[i].x = -GPU_MAX_BOUND.x;
			normal.x -= 1;
		}
		if (!GPU_TOP_OPEN)
		{
			if (x[i].y > GPU_MAX_BOUND.y)
			{
				x[i].y = GPU_MAX_BOUND.y;
				normal.y += 1;
			}
		}
		if (x[i].y < -GPU_MAX_BOUND.y)
		{
			x[i].y = -GPU_MAX_BOUND.y;
			normal.y -= 1;
		}
		if (x[i].z > GPU_MAX_BOUND.z)
		{
			x[i].z = GPU_MAX_BOUND.z;
			normal.z += 1;
		}
		if (x[i].z < -GPU_MAX_BOUND.z)
		{
			x[i].z = -GPU_MAX_BOUND.z;
			normal.z -= 1;
		}
		if (length(normal) > std::numeric_limits<float>::epsilon())
		{
			normal = normalize(normal);
			constexpr float c_f = 0.5f;
			v[i] -= (1.f + c_f) * dot(v[i], normal) * normal;
		}
	});
}

//void HinaPE::CUDA::DFSPH::solve(float dt)
//{
//
//	// ==================== 1. Build Neighbors ====================
//	if (need_reload)
//	{
//		Searcher->resize_point_set(fluid_idx, &(Fluid->x.data()->x), size);
//		need_reload = false;
//	}
//	Searcher->update_point_set(fluid_idx);
//	Searcher->find_neighbors();
//	cuNSearch::PointSet::NeighborSet &neighbor_set = Searcher->point_set(fluid_idx).get_raw_neighbor_set(fluid_idx);
//
//
//
//	// ==================== 2. Compute Density and Factor ====================
//	thrust::for_each(
//			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
//			[
//					x = Fluid->x.data(),
//					V = Fluid->V.data(),
//					rho = Fluid->rho.data(),
//					factor = Fluid->factor.data(),
//					nn = Fluid->nn.data(),
//					dNeighbors = neighbor_set.d_Neighbors.data(),
//					dCounts = neighbor_set.d_NeighborCounts.data(),
//					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
//			] __device__(size_t i)
//			{
//				float _rho_i = V[i] * W0();
//				float _sum_grad_p_k = 0;
//				float3 _grad_p_i = {0.f, 0.f, 0.f};
//				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
//				{
//					uint j = dNeighbors[dOffsets[i] + _n_idx];
//					_rho_i += V[j] * W(x[i] - x[j]);
//
//					float3 _grad_p_j = -V[j] * gradW(x[i] - x[j]);
//					_sum_grad_p_k += dot(_grad_p_j, _grad_p_j);
//					_grad_p_i -= _grad_p_j;
//				}
//
//				_sum_grad_p_k += dot(_grad_p_i, _grad_p_i);
//				rho[i] = _rho_i * REST_DENSITY;
//				if (_sum_grad_p_k > 1e-6f)
//					factor[i] = -1.f / _sum_grad_p_k;
//				else
//					factor[i] = 0;
//
//				nn[i] = dCounts[i];
//			});
//
//
//
//	// ==================== 3. Divergence Solve ====================
//	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
//	_) { return _ / dt; });
//	thrust::for_each(
//			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
//			[
//					x = Fluid->x.data(),
//					v = Fluid->v.data(),
//					V = Fluid->V.data(),
//					density_adv = Fluid->density_adv.data(),
//					dNeighbors = neighbor_set.d_Neighbors.data(),
//					dCounts = neighbor_set.d_NeighborCounts.data(),
//					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
//			] __device__(size_t i)
//			{
//				float _d_adv = 0;
//				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
//				{
//					uint j = dNeighbors[dOffsets[i] + _n_idx];
//					_d_adv += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
//				}
//				density_adv[i] = max(_d_adv, 0.f);
//			});
//	uint iteration_v = 0;
//	float avg_density_error = 0;
//	while (iteration_v < 1 || iteration_v < 100)
//	{
//		thrust::for_each(
//				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
//				[
//						x = Fluid->x.data(),
//						v = Fluid->v.data(),
//						V = Fluid->V.data(),
//						factor = Fluid->factor.data(),
//						density_adv = Fluid->density_adv.data(),
//						dNeighbors = neighbor_set.d_Neighbors.data(),
//						dCounts = neighbor_set.d_NeighborCounts.data(),
//						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
//						dt
//				] __device__(size_t i)
//				{
//					float _k_i = density_adv[i] * factor[i];
//					float3 _dv = {0.f, 0.f, 0.f};
//					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
//					{
//						uint j = dNeighbors[dOffsets[i] + _n_idx];
//						float _k_j = density_adv[j] * factor[j];
//						float _k_sum = _k_i + _k_j;
//						if (_k_sum > 1e-5f || _k_sum < -1e-5f)
//						{
//							float3 _grad_p_j = -V[j] * gradW(x[i] - x[j]);
//							_dv -= dt * _k_sum * _grad_p_j;
//						}
//					}
//					v[i] += _dv;
//				});
//		thrust::for_each(
//				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
//				[
//						x = Fluid->x.data(),
//						v = Fluid->v.data(),
//						V = Fluid->V.data(),
//						density_adv = Fluid->density_adv.data(),
//						dNeighbors = neighbor_set.d_Neighbors.data(),
//						dCounts = neighbor_set.d_NeighborCounts.data(),
//						dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
//				] __device__(size_t i)
//				{
//					float _d_adv = 0;
//					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
//					{
//						uint j = dNeighbors[dOffsets[i] + _n_idx];
//						_d_adv += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
//					}
//					density_adv[i] = max(_d_adv, 0.f);
//				});
//		thrust::transform(Fluid->density_adv.begin(), Fluid->density_adv.end(), Fluid->tmp.begin(), [] __device__(float _) { return _ * REST_DENSITY; });
//		avg_density_error = thrust::reduce(Fluid->tmp.begin(), Fluid->tmp.end(), 0.f, thrust::plus<float>()) / size;
//
//		const float eta = 1.f / dt * 0.1f * 0.01 * REST_DENSITY;
//		if (avg_density_error < eta)
//			break;
//		++iteration_v;
//	}
//	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
//	_) { return _ * dt; });
////	std::cout << "DFSPH - iteration V: " << iteration_v << " Avg density err: " << avg_density_error << std::endl;
//
//
//
//	// ==================== 4. Non-Pressure Force and Predict Velocity ====================
//	thrust::for_each(
//			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
//			[
//					x = Fluid->x.data(),
//					v = Fluid->v.data(),
//					a = Fluid->a.data(),
//					m = Fluid->m.data(),
//					rho = Fluid->rho.data(),
//					dNeighbors = neighbor_set.d_Neighbors.data(),
//					dCounts = neighbor_set.d_NeighborCounts.data(),
//					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
//			] __device__(size_t i)
//			{
//				float3 _dv = {0, -9.8f, 0};
//				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
//				{
//					uint j = dNeighbors[dOffsets[i] + _n_idx];
//					const float3 _r = x[i] - x[j];
//					const float _r2 = dot(_r, _r);
//					const float _r1 = sqrtf(_r2);
//					const float _diameter = PARTICLE_RADIUS * 2;
//					const float _diameter2 = _diameter * _diameter;
//					if (_r2 > _diameter2)
//						_dv -= SURFACE_TENSION / m[i] * m[j] * _r * W(_r1);
//					else
//						_dv -= SURFACE_TENSION / m[i] * m[j] * _r * W(_diameter);
//
////					float _v_xy = dot(v[i] - v[j], _r);
////					float3 _f_v = 10.f * VISCOSITY * (m[j] / rho[j]) * _v_xy / (_r2 + 0.01f * KERNEL_RADIUS * KERNEL_RADIUS) * gradW(_r);
////					_dv += _f_v;
//				}
//				a[i] = _dv;
//			});
//	thrust::transform(Fluid->a.begin(), Fluid->a.end(), Fluid->v.begin(), Fluid->v.begin(),[dt] __device__(float3
//	a, float3
//	v) { return v + dt * a; });
//
//
//
//	// ==================== 5. Pressure Solve ====================
//	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
//	_) { return _ / (dt * dt); });
//	thrust::for_each(
//			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
//			[
//					x = Fluid->x.data(),
//					v = Fluid->v.data(),
//					V = Fluid->V.data(),
//					rho = Fluid->rho.data(),
//					density_adv = Fluid->density_adv.data(),
//					dNeighbors = neighbor_set.d_Neighbors.data(),
//					dCounts = neighbor_set.d_NeighborCounts.data(),
//					dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
//					dt
//			] __device__(size_t i)
//			{
//				float _delta = 0;
//				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
//				{
//					uint j = dNeighbors[dOffsets[i] + _n_idx];
//					_delta += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
//				}
//				float _d_adv = rho[i] / REST_DENSITY + dt * _delta;
//				density_adv[i] = max(_d_adv, 1.f);
//			});
//	uint iteration_d = 0;
//	avg_density_error = 0;
//	while (iteration_d < 1 || iteration_d < 100)
//	{
//		thrust::for_each(
//				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
//				[
//						x = Fluid->x.data(),
//						v = Fluid->v.data(),
//						V = Fluid->V.data(),
//						factor = Fluid->factor.data(),
//						density_adv = Fluid->density_adv.data(),
//						dNeighbors = neighbor_set.d_Neighbors.data(),
//						dCounts = neighbor_set.d_NeighborCounts.data(),
//						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
//						dt
//				] __device__(size_t i)
//				{
//					float _k_i = (density_adv[i] - 1.f) * factor[i];
//					float3 _dv = {0.f, 0.f, 0.f};
//					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
//					{
//						uint j = dNeighbors[dOffsets[i] + _n_idx];
//						float _k_j = (density_adv[j] - 1.f) * factor[j];
//						float _k_sum = _k_i + _k_j;
//						if (_k_sum > 1e-5f || _k_sum < -1e-5f)
//						{
//							float3 _grad_p_j = -V[j] * gradW(x[i] - x[j]);
//							_dv -= dt * _k_sum * _grad_p_j;
//						}
//					}
//					v[i] += _dv;
//				});
//		thrust::for_each(
//				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
//				[
//						x = Fluid->x.data(),
//						v = Fluid->v.data(),
//						V = Fluid->V.data(),
//						rho = Fluid->rho.data(),
//						density_adv = Fluid->density_adv.data(),
//						dNeighbors = neighbor_set.d_Neighbors.data(),
//						dCounts = neighbor_set.d_NeighborCounts.data(),
//						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
//						dt
//				] __device__(size_t i)
//				{
//					float _delta = 0;
//					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
//					{
//						uint j = dNeighbors[dOffsets[i] + _n_idx];
//						_delta += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
//					}
//					float _d_adv = rho[i] / REST_DENSITY + dt * _delta;
//					density_adv[i] = max(_d_adv, 1.f);
//				});
////		avg_density_error = 0;
////		for (int iter = 0; iter < size; ++iter)
////			avg_density_error += REST_DENSITY * (Fluid->density_adv[iter] - 1.f);
////		avg_density_error /= size;
//
//		thrust::transform(Fluid->density_adv.begin(), Fluid->density_adv.end(), Fluid->tmp.begin(), [] __device__(float _) { return (_ - 1.f) * REST_DENSITY; });
//		avg_density_error = thrust::reduce(Fluid->tmp.begin(), Fluid->tmp.end(), 0.f, thrust::plus<float>()) / size;
//
//		const float eta = 0.05f * 0.01f * REST_DENSITY;
//		if (avg_density_error < eta)
//			break;
//		++iteration_d;
//	}
//	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
//	_) { return _ * (dt * dt); });
////	std::cout << "DFSPH - iteration: " << iteration_d << " Avg density err: " << avg_density_error << std::endl;
//
//
//
//	// ==================== 6. Advection ====================
//	thrust::transform(Fluid->v.begin(), Fluid->v.end(), Fluid->x.begin(), Fluid->x.begin(),[dt] __device__(float3
//	v, float3
//	x) { return x + dt * v; });
//
//
//
//	// ==================== 6. Boundary ====================
//	thrust::for_each(
//			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
//			[
//					x = Fluid->x.data(),
//					v = Fluid->v.data()
//			] __device__(size_t i)
//			{
//				float3 collision_normal{0, 0, 0};
//				if (x[i].x > MAX_BOUND.x)
//				{
//					x[i].x = MAX_BOUND.x;
//					collision_normal.x += 1;
//				}
//				if (x[i].x < -MAX_BOUND.x)
//				{
//					x[i].x = -MAX_BOUND.x;
//					collision_normal.x -= 1;
//				}
//				if (!TOP_OPEN)
//				{
//					if (x[i].y > MAX_BOUND.y)
//					{
//						x[i].y = MAX_BOUND.y;
//						collision_normal.y += 1;
//					}
//				}
//				if (x[i].y < -MAX_BOUND.y)
//				{
//					x[i].y = -MAX_BOUND.y;
//					collision_normal.y -= 1;
//				}
//				if (x[i].z > MAX_BOUND.z)
//				{
//					x[i].z = MAX_BOUND.z;
//					collision_normal.z += 1;
//				}
//				if (x[i].z < -MAX_BOUND.z)
//				{
//					x[i].z = -MAX_BOUND.z;
//					collision_normal.z -= 1;
//				}
//				if (length(collision_normal) > std::numeric_limits<float>::epsilon())
//				{
//					collision_normal = normalize(collision_normal);
//					v[i] -= (1. + 0.5f) * dot(v[i], collision_normal) * collision_normal;
//					v[i] *= 0.9f;
//				}
//			});
//}

#ifdef TEST_DFSPH
#include <vector>
#include <numeric>
int main()
{
	using Real = float;
	using Real3 = float3;
	std::vector<Real3> positions;
	std::size_t const N = 120;
	Real const r_omega = static_cast<Real>(0.15);
	Real const r_omega2 = r_omega * r_omega;
	Real const radius = static_cast<Real>(2.0) * (static_cast<Real>(2.0) * r_omega / static_cast<Real>(N - 1));

//Generate test data
	Real min_x = std::numeric_limits<Real>::max();
	Real max_x = std::numeric_limits<Real>::min();
	positions.reserve(N * N * N);
	for (uint i = 0; i < N; ++i)
	{
		for (uint j = 0; j < N; ++j)
		{
			for (uint k = 0; k < N; ++k)
			{
				std::array<Real, 3> x = {{
												 r_omega * static_cast<Real>(2.0 * static_cast<double>(i) / static_cast<double>(N - 1) - 1.0),
												 r_omega * static_cast<Real>(2.0 * static_cast<double>(j) / static_cast<double>(N - 1) - 1.0),
												 r_omega * static_cast<Real>(2.0 * static_cast<double>(k) / static_cast<double>(N - 1) - 1.0)}};

				Real l2 = x[0] * x[0] + x[1] * x[1] + x[2] * x[2];
				if (l2 < r_omega2)
				{
					x[0] += static_cast<Real>(0.35);
					x[1] += static_cast<Real>(0.35);
					x[2] += static_cast<Real>(0.35);
					positions.push_back(make_float3(x[0], x[1], x[2]));
					if (min_x > x[0])
					{
						min_x = x[0];
					}
					if (max_x < x[0])
					{
						max_x = x[0];
					}
				}
			}
		}
	}
	printf("Number of particles: %d \n", static_cast<int>(positions.size()));

	//Create neighborhood search instance
	cuNSearch::NeighborhoodSearch nsearch(radius);

	//Add point set from the test data
	auto pointSetIndex = nsearch.add_point_set(&positions.front().x, positions.size(), true, true);

	for (size_t i = 0; i < 5; i++)
	{
		if (i != 0)
		{
			nsearch.z_sort();
			nsearch.point_set(pointSetIndex).sort_field((Real3 *) nsearch.point_set(pointSetIndex).GetPoints());
		}

		nsearch.find_neighbors();
	}

	//Neighborhood search result test
	auto &pointSet = nsearch.point_set(0);
	auto points = pointSet.GetPoints();

	std::cout << "Validate results" << std::endl;
	for (uint i = 0; i < pointSet.n_points(); i++)
	{
		Real3 point = ((Real3 *) points)[i];
		auto count = pointSet.n_neighbors(0, i);
		for (uint j = 0; j < count; j++)
		{
			auto neighbor = pointSet.neighbor(0, i, j);
			Real3 diff = {point.x - ((Real3 *) points)[neighbor].x, point.y - ((Real3 *) points)[neighbor].y, point.z - ((Real3 *) points)[neighbor].z};
			float squaredLength = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
			float distance = sqrt(squaredLength);

			if (distance > radius)
			{
				throw std::runtime_error("Not a neighbor");
			}
		}
	}
	return 0;
}
#endif
