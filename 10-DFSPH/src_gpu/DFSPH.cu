#include "hip/hip_runtime.h"
#include "DFSPH.cuh"
#include <hip/hip_runtime.h>
#include "common/hip/hip_runtime_api.h"
#include "common/hip/hip_vector_types.h"
#include "neighbor/include/cuNSearch.h"
#include "neighbor/src/cuNSearchDeviceData.h"
#include <thrust/execution_policy.h>

__constant__ float PARTICLE_RADIUS;
__constant__ float KERNEL_RADIUS;
__constant__ float KERNEL_K;
__constant__ float KERNEL_L;
__constant__ float REST_DENSITY;
__constant__ float VISCOSITY;
__constant__ float SURFACE_TENSION;
__constant__ float3 MAX_BOUND;
__constant__ bool TOP_OPEN;
static inline __device__ float W(const float r)
{
	float res = 0.f;
	const float q = r / KERNEL_RADIUS;
	if (q <= 1.f)
	{
		if (q <= 0.5f)
		{
			const float q2 = q * q;
			const float q3 = q2 * q;
			res = KERNEL_K * (6.f * q3 - 6.f * q2 + 1.f);
		} else
		{
			res = KERNEL_K * (2.f * powf(1.f - q, 3.f));
		}
	}
	return res;
}
static inline __device__ float W(const float3 r) { return W(length(r)); }
static inline __device__ float3 gradW(const float3 r)
{
	float3 res = {0.f, 0.f, 0.f};
	const float rl = length(r);
	const float q = rl / KERNEL_RADIUS;
	if ((rl > 1.0e-9) && (q <= 1.f))
	{
		float3 gradq = {r.x / rl, r.y / rl, r.z / rl};
		gradq.x /= KERNEL_RADIUS;
		gradq.y /= KERNEL_RADIUS;
		gradq.z /= KERNEL_RADIUS;
		if (q <= 0.5f)
		{
			res = KERNEL_L * q * (3.f * q - 2.f) * gradq;
		} else
		{
			const float factor = 1.f - q;
			res = KERNEL_L * (-factor * factor) * gradq;
		}
	}
	return res;
}
static inline __device__ float W0() { return W(make_float3(0.f, 0.f, 0.f)); }

HinaPE::CUDA::DFSPH::DFSPH(float _kernel_radius) : size(0)
{
	float kr = _kernel_radius;
	float r = 0.01f;
	float k = 8.f / (3.14159265358979323846f * kr * kr * kr);
	float l = 48.f / (3.14159265358979323846f * kr * kr * kr);
	float rd = 1000.f;
	float vis = 0.01f;
	float st = 0.01f;
	float3 max_bound = {2.f, 2.f, 2.f};
	bool top_open = true;
	hipMalloc((void **) &PARTICLE_RADIUS, sizeof(float));
	hipMalloc((void **) &KERNEL_RADIUS, sizeof(float));
	hipMalloc((void **) &KERNEL_K, sizeof(float));
	hipMalloc((void **) &KERNEL_L, sizeof(float));
	hipMalloc((void **) &REST_DENSITY, sizeof(float));
	hipMalloc((void **) &VISCOSITY, sizeof(float));
	hipMalloc((void **) &SURFACE_TENSION, sizeof(float));
	hipMalloc((void **) &MAX_BOUND, sizeof(float3));
	hipMalloc((void **) &TOP_OPEN, sizeof(bool));
	hipMemcpyToSymbol(HIP_SYMBOL(PARTICLE_RADIUS), &r, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_RADIUS), &kr, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_K), &k, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_L), &l, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(REST_DENSITY), &rd, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(VISCOSITY), &vis, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(SURFACE_TENSION), &st, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(MAX_BOUND), &max_bound, 3 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(TOP_OPEN), &top_open, sizeof(bool));

	Fluid = std::make_shared<FluidGPU>();
	Searcher = std::make_shared<cuNSearch::NeighborhoodSearch>(kr);
	fluid_idx = Searcher->add_point_set(&(Fluid->x.data()->x), Fluid->x.size(), true, true, true);
	Searcher->set_active(true);
}

void HinaPE::CUDA::DFSPH::resize(size_t n)
{
	if (size == n)
		return;
	Fluid->x.resize(n, make_float3(0, 0, 0));
	Fluid->v.resize(n, make_float3(0, 0, 0));
	Fluid->a.resize(n, make_float3(0, 0, 0));
	Fluid->m.resize(n, 1000.f * 0.9f * 0.02f * 0.02f * 0.02f);
	Fluid->V.resize(n, 0.9f * 0.02f * 0.02f * 0.02f);
	Fluid->rho.resize(n, 0);
	Fluid->factor.resize(n, 0);
	Fluid->density_adv.resize(n, 0);
	Fluid->nn.resize(n, 0);
	Fluid->tmp.resize(n, 0);
	size = n;

	need_reload = true;
}

void HinaPE::CUDA::DFSPH::solve(float dt)
{

	// ==================== 1. Build Neighbors ====================
	if (need_reload)
	{
		Searcher->resize_point_set(fluid_idx, &(Fluid->x.data()->x), size);
		need_reload = false;
	}
	Searcher->update_point_set(fluid_idx);
	Searcher->find_neighbors();
	cuNSearch::PointSet::NeighborSet &neighbor_set = Searcher->point_set(fluid_idx).get_raw_neighbor_set(fluid_idx);



	// ==================== 2. Compute Density and Factor ====================
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					V = Fluid->V.data(),
					rho = Fluid->rho.data(),
					factor = Fluid->factor.data(),
					nn = Fluid->nn.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
			] __device__(size_t i)
			{
				float _rho_i = V[i] * W0();
				float _sum_grad_p_k = 0;
				float3 _grad_p_i = {0.f, 0.f, 0.f};
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];
					_rho_i += V[j] * W(x[i] - x[j]);

					float3 _grad_p_j = -V[j] * gradW(x[i] - x[j]);
					_sum_grad_p_k += dot(_grad_p_j, _grad_p_j);
					_grad_p_i -= _grad_p_j;
				}

				_sum_grad_p_k += dot(_grad_p_i, _grad_p_i);
				rho[i] = _rho_i * REST_DENSITY;
				if (_sum_grad_p_k > 1e-6f)
					factor[i] = -1.f / _sum_grad_p_k;
				else
					factor[i] = 0;

				nn[i] = dCounts[i];
			});



	// ==================== 3. Divergence Solve ====================
	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float _) { return _ / dt; });
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					V = Fluid->V.data(),
					density_adv = Fluid->density_adv.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
			] __device__(size_t i)
			{
				float _d_adv = 0;
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];
					_d_adv += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
				}
				density_adv[i] = max(_d_adv, 0.f);
			});
	uint iteration_v = 0;
	float avg_density_error = 0;
	while (iteration_v < 1 || iteration_v < 100)
	{
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						factor = Fluid->factor.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
						dt
				] __device__(size_t i)
				{
					float _k_i = density_adv[i] * factor[i];
					float3 _dv = {0.f, 0.f, 0.f};
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						float _k_j = density_adv[j] * factor[j];
						float _k_sum = _k_i + _k_j;
						if (_k_sum > 1e-5f || _k_sum < -1e-5f)
						{
							float3 _grad_p_j = -V[j] * gradW(x[i] - x[j]);
							_dv -= dt * _k_sum * _grad_p_j;
						}
					}
					v[i] += _dv;
				});
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
				] __device__(size_t i)
				{
					float _d_adv = 0;
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						_d_adv += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
					}
					density_adv[i] = max(_d_adv, 0.f);
				});
		thrust::transform(Fluid->density_adv.begin(), Fluid->density_adv.end(), Fluid->tmp.begin(), [] __device__(float _) { return _ * REST_DENSITY; });
		avg_density_error = thrust::reduce(Fluid->tmp.begin(), Fluid->tmp.end(), 0.f, thrust::plus<float>()) / size;

		const float eta = 1.f / dt * 0.1f * 0.01 * REST_DENSITY;
		if (avg_density_error < eta)
			break;
		++iteration_v;
	}
	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
	_) { return _ * dt; });
//	std::cout << "DFSPH - iteration V: " << iteration_v << " Avg density err: " << avg_density_error << std::endl;



	// ==================== 4. Non-Pressure Force and Predict Velocity ====================
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					a = Fluid->a.data(),
					m = Fluid->m.data(),
					rho = Fluid->rho.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data()
			] __device__(size_t i)
			{
				float3 _dv = {0, -9.8f, 0};
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];
					const float3 _r = x[i] - x[j];
					const float _r2 = dot(_r, _r);
					const float _r1 = sqrtf(_r2);
					const float _diameter = PARTICLE_RADIUS * 2;
					const float _diameter2 = _diameter * _diameter;
					if (_r2 > _diameter2)
						_dv -= SURFACE_TENSION / m[i] * m[j] * _r * W(_r1);
					else
						_dv -= SURFACE_TENSION / m[i] * m[j] * _r * W(_diameter);

//					float _v_xy = dot(v[i] - v[j], _r);
//					float3 _f_v = 10.f * VISCOSITY * (m[j] / rho[j]) * _v_xy / (_r2 + 0.01f * KERNEL_RADIUS * KERNEL_RADIUS) * gradW(_r);
//					_dv += _f_v;
				}
				a[i] = _dv;
			});
	thrust::transform(Fluid->a.begin(), Fluid->a.end(), Fluid->v.begin(), Fluid->v.begin(),[dt] __device__(float3
	a, float3
	v) { return v + dt * a; });



	// ==================== 5. Pressure Solve ====================
	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
	_) { return _ / (dt * dt); });
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data(),
					V = Fluid->V.data(),
					rho = Fluid->rho.data(),
					density_adv = Fluid->density_adv.data(),
					dNeighbors = neighbor_set.d_Neighbors.data(),
					dCounts = neighbor_set.d_NeighborCounts.data(),
					dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
					dt
			] __device__(size_t i)
			{
				float _delta = 0;
				for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
				{
					uint j = dNeighbors[dOffsets[i] + _n_idx];
					_delta += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
				}
				float _d_adv = rho[i] / REST_DENSITY + dt * _delta;
				density_adv[i] = max(_d_adv, 1.f);
			});
	uint iteration_d = 0;
	avg_density_error = 0;
	while (iteration_d < 1 || iteration_d < 100)
	{
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						factor = Fluid->factor.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
						dt
				] __device__(size_t i)
				{
					float _k_i = (density_adv[i] - 1.f) * factor[i];
					float3 _dv = {0.f, 0.f, 0.f};
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						float _k_j = (density_adv[j] - 1.f) * factor[j];
						float _k_sum = _k_i + _k_j;
						if (_k_sum > 1e-5f || _k_sum < -1e-5f)
						{
							float3 _grad_p_j = -V[j] * gradW(x[i] - x[j]);
							_dv -= dt * _k_sum * _grad_p_j;
						}
					}
					v[i] += _dv;
				});
		thrust::for_each(
				thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
				[
						x = Fluid->x.data(),
						v = Fluid->v.data(),
						V = Fluid->V.data(),
						rho = Fluid->rho.data(),
						density_adv = Fluid->density_adv.data(),
						dNeighbors = neighbor_set.d_Neighbors.data(),
						dCounts = neighbor_set.d_NeighborCounts.data(),
						dOffsets = neighbor_set.d_NeighborWriteOffsets.data(),
						dt
				] __device__(size_t i)
				{
					float _delta = 0;
					for (uint _n_idx = 0; _n_idx < dCounts[i]; ++_n_idx)
					{
						uint j = dNeighbors[dOffsets[i] + _n_idx];
						_delta += V[j] * dot(v[i] - v[j], gradW(x[i] - x[j]));
					}
					float _d_adv = rho[i] / REST_DENSITY + dt * _delta;
					density_adv[i] = max(_d_adv, 1.f);
				});
//		avg_density_error = 0;
//		for (int iter = 0; iter < size; ++iter)
//			avg_density_error += REST_DENSITY * (Fluid->density_adv[iter] - 1.f);
//		avg_density_error /= size;

		thrust::transform(Fluid->density_adv.begin(), Fluid->density_adv.end(), Fluid->tmp.begin(), [] __device__(float _) { return (_ - 1.f) * REST_DENSITY; });
		avg_density_error = thrust::reduce(Fluid->tmp.begin(), Fluid->tmp.end(), 0.f, thrust::plus<float>()) / size;

		const float eta = 0.05f * 0.01f * REST_DENSITY;
		if (avg_density_error < eta)
			break;
		++iteration_d;
	}
	thrust::transform(Fluid->factor.begin(), Fluid->factor.end(), Fluid->factor.begin(),[dt] __device__(float
	_) { return _ * (dt * dt); });
//	std::cout << "DFSPH - iteration: " << iteration_d << " Avg density err: " << avg_density_error << std::endl;



	// ==================== 6. Advection ====================
	thrust::transform(Fluid->v.begin(), Fluid->v.end(), Fluid->x.begin(), Fluid->x.begin(),[dt] __device__(float3 v, float3 x) { return x + dt * v; });



	// ==================== 6. Boundary ====================
	thrust::for_each(
			thrust::make_counting_iterator((size_t) 0), thrust::make_counting_iterator(size),
			[
					x = Fluid->x.data(),
					v = Fluid->v.data()
			] __device__(size_t i)
			{
				float3 collision_normal{0, 0, 0};
				if (x[i].x > MAX_BOUND.x)
				{
					x[i].x = MAX_BOUND.x;
					collision_normal.x += 1;
				}
				if (x[i].x < -MAX_BOUND.x)
				{
					x[i].x = -MAX_BOUND.x;
					collision_normal.x -= 1;
				}
				if (!TOP_OPEN)
				{
					if (x[i].y > MAX_BOUND.y)
					{
						x[i].y = MAX_BOUND.y;
						collision_normal.y += 1;
					}
				}
				if (x[i].y < -MAX_BOUND.y)
				{
					x[i].y = -MAX_BOUND.y;
					collision_normal.y -= 1;
				}
				if (x[i].z > MAX_BOUND.z)
				{
					x[i].z = MAX_BOUND.z;
					collision_normal.z += 1;
				}
				if (x[i].z < -MAX_BOUND.z)
				{
					x[i].z = -MAX_BOUND.z;
					collision_normal.z -= 1;
				}
				if (length(collision_normal) > std::numeric_limits<float>::epsilon())
				{
					collision_normal = normalize(collision_normal);
					v[i] -= (1. + 0.5f) * dot(v[i], collision_normal) * collision_normal;
					v[i] *= 0.9f;
				}
			});
}

#ifdef TEST_DFSPH
#include <vector>
#include <numeric>
int main()
{
	using Real = float;
	using Real3 = float3;
	std::vector<Real3> positions;
	std::size_t const N = 120;
	Real const r_omega = static_cast<Real>(0.15);
	Real const r_omega2 = r_omega * r_omega;
	Real const radius = static_cast<Real>(2.0) * (static_cast<Real>(2.0) * r_omega / static_cast<Real>(N - 1));

//Generate test data
	Real min_x = std::numeric_limits<Real>::max();
	Real max_x = std::numeric_limits<Real>::min();
	positions.reserve(N * N * N);
	for (uint i = 0; i < N; ++i)
	{
		for (uint j = 0; j < N; ++j)
		{
			for (uint k = 0; k < N; ++k)
			{
				std::array<Real, 3> x = {{
												 r_omega * static_cast<Real>(2.0 * static_cast<double>(i) / static_cast<double>(N - 1) - 1.0),
												 r_omega * static_cast<Real>(2.0 * static_cast<double>(j) / static_cast<double>(N - 1) - 1.0),
												 r_omega * static_cast<Real>(2.0 * static_cast<double>(k) / static_cast<double>(N - 1) - 1.0)}};

				Real l2 = x[0] * x[0] + x[1] * x[1] + x[2] * x[2];
				if (l2 < r_omega2)
				{
					x[0] += static_cast<Real>(0.35);
					x[1] += static_cast<Real>(0.35);
					x[2] += static_cast<Real>(0.35);
					positions.push_back(make_float3(x[0], x[1], x[2]));
					if (min_x > x[0])
					{
						min_x = x[0];
					}
					if (max_x < x[0])
					{
						max_x = x[0];
					}
				}
			}
		}
	}
	printf("Number of particles: %d \n", static_cast<int>(positions.size()));

	//Create neighborhood search instance
	cuNSearch::NeighborhoodSearch nsearch(radius);

	//Add point set from the test data
	auto pointSetIndex = nsearch.add_point_set(&positions.front().x, positions.size(), true, true);

	for (size_t i = 0; i < 5; i++)
	{
		if (i != 0)
		{
			nsearch.z_sort();
			nsearch.point_set(pointSetIndex).sort_field((Real3 *) nsearch.point_set(pointSetIndex).GetPoints());
		}

		nsearch.find_neighbors();
	}

	//Neighborhood search result test
	auto &pointSet = nsearch.point_set(0);
	auto points = pointSet.GetPoints();

	std::cout << "Validate results" << std::endl;
	for (uint i = 0; i < pointSet.n_points(); i++)
	{
		Real3 point = ((Real3 *) points)[i];
		auto count = pointSet.n_neighbors(0, i);
		for (uint j = 0; j < count; j++)
		{
			auto neighbor = pointSet.neighbor(0, i, j);
			Real3 diff = {point.x - ((Real3 *) points)[neighbor].x, point.y - ((Real3 *) points)[neighbor].y, point.z - ((Real3 *) points)[neighbor].z};
			float squaredLength = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
			float distance = sqrt(squaredLength);

			if (distance > radius)
			{
				throw std::runtime_error("Not a neighbor");
			}
		}
	}
	return 0;
}
#endif
